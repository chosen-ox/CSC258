/*
    Created by: Andrew Sexton
          Date: March 21, 2022
      Modified: February 20, 2023 by Owen Wacha

    CSC258/458 - Parallel & Distributed Systems.
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cmath>
#include <stdlib.h>
#include <time.h>

/* Use this macro to catch and print out runtime errors from the GPU */
/* This does not work on kernel functions eg. some_kernel<<<...>>>() */
/* Ex. cudaErrChk(cudaMalloc(...)) */
/*     cudaErrChk(cudaDeviceSynchronize()) */
#define cudaErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true) {
    if (code != hipSuccess) {
        std::cout << "GPUAssert: " << hipGetErrorString(code) << " " << file << " line " << line << std::endl;
        if (abort) { exit(code); }
    }
}

/* Vectorizable version of matrix multiplication for comparison */
__global__ void seq_matmul(const float* A, const float* B, float* C, int nsize) {
    float temp;
    for (int i = 0; i < nsize; i++) {
        for (int j = 0; j < nsize; j++) {
            temp = 0.0f;
            for (int k = 0; k < nsize; k++) {
                temp += A[k + (i * nsize)] * B[j + (k * nsize)];
            }
            C[j + (i * nsize)] = temp;
        }
    }
}

// Function for verifying values between two arrays
// by computing abs(X[i] - Y[i]) < EPSILON
void verify(const float* X, const float* Y, int nsize){
    float EPSILON = 1E-4;
    for(int i = 0; i < nsize; i++) {
        for(int j = 0; j < nsize; j++) {
            int idx = j + (i * nsize);

            if(std::fabs(X[idx] - Y[idx]) > EPSILON) {
                std::cout << std::setprecision(15) << "(" << i << ", " << j << "): " << X[idx] << " != " << Y[idx] << std::endl;
            }
        }
    }
}

// Print a comma-separated 2D array to stdout
void print_array(const float* arr, int nsize) {
    for(int i = 0; i < nsize; i++) {
        for(int j = 0; j < nsize; j++) {
            std::cout << arr[j + (i * nsize)];

            if(j < nsize) {
                std::cout << ", ";
            }
        }
        std::cout << std::endl;
    }
}

// GPU Kernel
__global__ void gpu_matmul(float* A, float* B, float* C, int nsize) {
    /* Add your cuda solution code here */
    //
    // int x = blockIdx.x * blockDim.x + threadIdx.x;
    // int y = blockIdx.y * blockDim.y + threadIdx.y;
    // if (x < nsize && y < nsize) {
    // float tmp = 0.0f;
    // for(int k = 0; k < nsize; k++) {
    //     tmp += A[k + (x * nsize)] * B[y + (k * nsize)]; 
    // }

    // C[x*nsize + y] = tmp; 
    // }
    //

    //
int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if( col < nsize && row < nsize) 
    {
        for(int i = 0; i < nsize; i++) 
        {
            sum += A[row * nsize + i] * B[i * nsize + col];
        }
        C[row * nsize + col] = sum;
    }

}


int main(int argc, char *argv[]) {
    if(argc < 2) {
        std::cout << "Invalid number of arguments: usage " << argv[0] << " <array size>" << std::endl;
        exit(0);
    }

    // Array size
    int nsize = std::atoi(argv[1]);

    // Timing Stuff
    timespec seq_start, seq_stop;
    timespec gpu_start, gpu_stop;

    // CPU side arrays
    // Arrays are one dimensional, indexing is (i, j) => j + (i * nsize)
    // this gives a single index into the array using two loop variables
    float* A = new float[nsize * nsize]();
    float* B = new float[nsize * nsize]();
    float* C = new float[nsize * nsize]();
    float* D = new float[nsize * nsize]();

    float * A1;
    float * B1;
    float * C1;
    float * D1;
    size_t bytes = nsize * nsize * sizeof(float);

    // cudaMalloc(&A1, bytes);
    // cudaMalloc(&B1, bytes);
    // cudaMalloc(&C1, bytes);

        cudaErrchk(hipMalloc(&A1, bytes));
        cudaErrchk(hipMalloc(&B1, bytes));
        cudaErrchk(hipMalloc(&C1, bytes));
        cudaErrchk(hipMalloc(&D1, bytes));


    // Fill CPU side arrays
    for(int i = 0; i < nsize; i++) {
        for(int j = 0; j < nsize; j++) {
            int idx = static_cast<float>(j + (i * nsize));
            A[idx] = idx + 1.0f;
            B[idx] = 1.0f / (idx + 1.0f);
        }
    }

    cudaErrchk(hipMemcpy( A1, A, bytes, hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy( B1, B, bytes, hipMemcpyHostToDevice));
    // Start GPU timer

    /* Add your code here */
    //
    //
    //
    /*===================*/
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(nsize/ threadsPerBlock.x + 1, nsize / threadsPerBlock.y + 1);
    clock_gettime(CLOCK_REALTIME, &gpu_start);
    gpu_matmul<<<numBlocks, threadsPerBlock>>>(A1, B1, C1, nsize);

    // gpu_matmul<<<numBlocks, threadsPerBlock>>>(A1, B1, C1, nsize);

        // cudaErrchk(cudaMemcpy(D, C1, bytes, cudaMemcpyDeviceToHost ));
        // gpuErrchk(cudaFree(DadC1,
        // gpuErrchk(cudaFree( bd ));
    // Stop GPU timer
    //clock_gettime(CLOCK_REALTIME, &gpu_stop);	
    cudaErrchk( hipPeekAtLastError() );
    cudaErrchk( hipDeviceSynchronize() );
    cudaErrchk(hipMemcpy( C, C1, bytes, hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME, &gpu_stop); 
    // cudaErrchk(cudaFree(A1));
    // cudaErrchk(cudaFree(B1));
    // cudaErrchk(cudaFree(C1));
    //print_array(C, nsize);
    std::cout << "GPU Time: " << ((gpu_stop.tv_sec - gpu_start.tv_sec) + (gpu_stop.tv_nsec - gpu_start.tv_nsec) / 1E9) << '\n';
    std::cout << "CUDA speed:" << ( (nsize * nsize)/ ((gpu_stop.tv_sec - gpu_stop.tv_sec) + (gpu_stop.tv_nsec - gpu_start.tv_nsec) / 1E9)  * nsize * 2 / 1E6 )<< '\n';

    // Compute Vectorized version
    // Modifies array C in place.
    clock_gettime(CLOCK_REALTIME, &seq_start);
    seq_matmul<<<1, 1>>>(A1, B1, D1, nsize);
    cudaErrchk( hipPeekAtLastError() );
    cudaErrchk( hipDeviceSynchronize() );
    // seq_matmul(A, B, C, nsize);
    clock_gettime(CLOCK_REALTIME, &seq_stop);
    printf("111\n");
    std::cout << "Seq (vectorized) Time: " << ((seq_stop.tv_sec - seq_start.tv_sec) + (seq_stop.tv_nsec - seq_start.tv_nsec) / 1E9) << '\n';
    cudaErrchk(hipMemcpy( D, D1, bytes, hipMemcpyDeviceToHost));
    // print_array(C, nsize);

    /* Add Verification Step Here */
    // ...
    // verify(C, D, nsize);

    // Clean up
    delete[] A;
    delete[] B;
    delete[] C;
    delete[] D;
    cudaErrchk(hipFree(A1));
    cudaErrchk(hipFree(B1));
    cudaErrchk(hipFree(C1));
    cudaErrchk(hipFree(D1));

    return 0;
}
